#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <stdio.h>
#include <cstdint>

#define DEBUG_1D 0
#define DEBUG_THREAD_INFO_FLOAT32 0
#define DEBUG_THREAD_INFO_INT32 0
#define DEBUG_BITS 0
#define DEBUG_SEEDS 0

template <typename scalar_t>
__global__ void customconv1d_kernel(
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> weight,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> output,
    torch::PackedTensorAccessor<scalar_t,1,torch::RestrictPtrTraits,size_t> threshold,
    int nr_xnor_gates
  )
{

  // handle access indices
  const int c = blockIdx.x * blockDim.x + threadIdx.x; // y
  const int d = blockIdx.y * blockDim.y + threadIdx.y; // x

  // make sure we don't modify memory regions outside of output
  if ((d < output.size(0)) && (c < output.size(1)))
  {
    // this is (c,d,e), we have as many threads as we have pixels in output out
    // each thread of out calculates a MAC (row of filter times column of input)

    // every thread is responsible for one sum, there are as many threads as mac sums in output
    output[d][c] = 0; // output buffer
    float result = 0;
    float sub_popcnt = 0; // used for sub-popcount computations
    int cycle_counter = 0; // nr of cycles the tlu has executed at present
    float cycles = weight.size(1) / nr_xnor_gates; // nr of cycles the tlu has to execute

    float threshold_for_sample = threshold[c] / cycles;
    int comparison = 0;

    // #if 1
    //   if (d == 0 && c == 1)
    //   {
    //     printf("cycles: %.2f, threshold: %.2f, threshold_sample: %2.f\n", cycles, threshold[c], threshold_for_sample);
    //   }
    // #endif

    for(int i = 0; i < weight.size(1); i++)
    {
      //printf("Thread: (%d,%d,%d)\nWeight: %.4f, Input: %.4f\n", c, d, e, weight[c][i], input[d][i][e]);
      sub_popcnt += weight[c][i] * input[d][i];
      cycle_counter += 1;

      if (cycle_counter == nr_xnor_gates)
      {
        comparison = (sub_popcnt <= threshold_for_sample);
        result += comparison;
        sub_popcnt = 0;
        cycle_counter = 0;
      }

      // // edge case
      // if (i == weight.size(1)-1)
      // {
      //   mult_result += sub_popcnt;
      //   // TODO: threshold for edge case round(((weight.size(1) % nr_xnor_gates) / nr_xnor_gates)*threshold[c])
      // }
    }

    if (result <= cycles/2)
    {
      output[d][c] = 1;
    }
    else
    {
      output[d][c] = -1;
    }
  }
}

torch::Tensor customconv1d_cuda(
  torch::Tensor input,
  torch::Tensor weight,
  torch::Tensor output,
  torch::Tensor threshold,
  int nr_xnor_gates
) {
  // The number of thread blocks in a grid is usually dictated by the size of the data being processed, which typically exceeds the number of processors in the system.
  // dim3 threadsPerBlock(8,8,8)
  // <<<number of blocks per grid, number of threads ber block>>>
  // grid is created with enough blocks to have one thread per matrix element

  // https://devtalk.nvidia.com/default/topic/1028226/how-many-concurrent-threads-are-running-on-my-geforce-gtx-1080-ti-/
  const int output_size_x = output.size(1);
  const int output_size_y = output.size(0);
  int threads_x = 16; // per block, 16
  int threads_y = 16; // per block, 16

  #if DEBUG_1D
    threads_x = 1;
    threads_y = 1;
  #endif

  const dim3 threads(threads_x,threads_y);
  const dim3 blocks((output_size_x + threads_x - 1) / threads_x,
                    (output_size_y + threads_y - 1) / threads_y);

  AT_DISPATCH_ALL_TYPES(input.type(), "customconv1d_cuda", ([&] {
    customconv1d_kernel<scalar_t><<<blocks, threads>>>(
        input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        weight.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        threshold.packed_accessor<scalar_t,1,torch::RestrictPtrTraits,size_t>(),
        nr_xnor_gates
    );
  }));

  return output;
}
