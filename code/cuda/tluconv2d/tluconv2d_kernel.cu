#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <stdio.h>
#include <cstdint>

#define DEBUG_1D 0
#define DEBUG_THREAD_INFO_FLOAT32 0
#define DEBUG_THREAD_INFO_INT32 0
#define DEBUG_BITS 0
#define DEBUG_SEEDS 0

template <typename scalar_t>
__global__ void customconv2d_kernel(
    torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> input,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> weight,
    torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> output,
    torch::PackedTensorAccessor<scalar_t,1,torch::RestrictPtrTraits,size_t> threshold,
    int nr_xnor_gates,
    int nr_additional_samples,
    int majv_shift,
    int threshold_scaling
  )
{

  // handle access indices
  const int c = blockIdx.x * blockDim.x + threadIdx.x; // y
  const int d = blockIdx.y * blockDim.y + threadIdx.y; // x
  const int e = blockIdx.z * blockDim.z + threadIdx.z; // z

  // make sure we don't modify memory regions outside of output
  if ((d < output.size(0)) && (c < output.size(1)) && (e < output.size(2)))
  {
    // this is (c,d,e), we have as many threads as we have pixels in output out
    // each thread of out calculates a MAC (row of filter times column of input)

    // every thread is responsible for one sum, there are as many threads as mac sums in output
    output[d][c][e] = 0; // output buffer
    float result = 0;
    float sub_popcnt = 0; // used for sub-popcount computations
    float sub_popcnt_1 = 0; // used for sub-popcount computations with one more sample
    float sub_popcnt_2_1 = 0; // used for sub-popcount computations with two more samples
    float sub_popcnt_2_2 = 0; // used for sub-popcount computations with two more samples
    int cycle_counter = 0; // nr of cycles the tlu has executed at present
    float cycles = weight.size(1) / nr_xnor_gates; // nr of cycles the tlu has to execute

    float threshold_for_sample = round(threshold[c] / cycles);
    if (threshold_scaling == 2)
    {
      threshold_for_sample = 2*floorf(threshold_for_sample/2);
    }
    float last_threshold_for_sample = 0;
    int comparison = 0;

    // #if 1
    //   if (d == 0 && c == 1)
    //   {
    //     printf("cycles: %.2f, threshold: %.2f, threshold_sample: %2.f\n", cycles, threshold[c], threshold_for_sample);
    //   }
    // #endif

    for(int i = 0; i < weight.size(1); i++)
    {
      //printf("Thread: (%d,%d,%d)\nWeight: %.4f, Input: %.4f\n", c, d, e, weight[c][i], input[d][i][e]);
      sub_popcnt += weight[c][i] * input[d][i][e];
      cycle_counter += 1;

      // one more sample, in the middle of two subsequent samples (overlapping)
      if (nr_additional_samples == 1)
      {
        if (i + nr_xnor_gates <= (weight.size(1) - 1))
        {
          sub_popcnt_1 += weight[c][i + (nr_xnor_gates / 2)] * input[d][i + (nr_xnor_gates / 2)][e];
        }
      }

      // two more samples
      if (nr_additional_samples == 2)
      {
        if (i + nr_xnor_gates <= (weight.size(1) - 1))
        {
          sub_popcnt_2_1 += weight[c][i + round((nr_xnor_gates)*(1/3))] * input[d][i + round((nr_xnor_gates)*(1/3))][e];
          sub_popcnt_2_2 += weight[c][i + round((nr_xnor_gates)*(2/3))] * input[d][i + round((nr_xnor_gates)*(2/3))][e];
        }
      }

      // when "nr_xnor_gates" many operations have been computed
      if (cycle_counter == nr_xnor_gates)
      {
        comparison = (sub_popcnt >= threshold_for_sample);
        result += comparison;
        sub_popcnt = 0;
        cycle_counter = 0;

        if (nr_additional_samples == 1)
        {
          comparison = (sub_popcnt_1 >= threshold_for_sample);
          result += comparison;
          sub_popcnt_1 = 0;
        }

        if (nr_additional_samples == 2)
        {
          comparison = (sub_popcnt_2_1 >= threshold_for_sample);
          result += comparison;
          sub_popcnt_2_1 = 0;

          comparison = (sub_popcnt_2_2 >= threshold_for_sample);
          result += comparison;
          sub_popcnt_2_2 = 0;
        }

      }

      // edge case
      if ((i == weight.size(1)-1)
          && ((weight.size(1) % nr_xnor_gates) != 0))
      {
        last_threshold_for_sample = round(((weight.size(1) % nr_xnor_gates) / nr_xnor_gates) * threshold[c]);
        comparison = (sub_popcnt >= last_threshold_for_sample);
        result += comparison;
      }
    }

    if (nr_additional_samples == 2)
    {
      // if (result <= round((cycles/2)*2*((cycles-1)/cycles)))
      if (result <= ((cycles/2) + 2*((cycles-1)/2) + majv_shift))
      {
        output[d][c][e] = -1;
      }
      else
      {
        output[d][c][e] = 1;
      }
    }

    if (nr_additional_samples == 1)
    {
      // if (result <= round((cycles/2)*2*((cycles-1)/cycles)))
      if (result <= ((cycles/2) + ((cycles-1)/2) + majv_shift))
      {
        output[d][c][e] = -1;
      }
      else
      {
        output[d][c][e] = 1;
      }
    }

    if (nr_additional_samples == 0)
    {
      if (result <= ((cycles/2) + majv_shift))
      {
        output[d][c][e] = -1;
      }
      else
      {
        output[d][c][e] = 1;
      }
    }
  }
}

torch::Tensor customconv2d_cuda(
  torch::Tensor input,
  torch::Tensor weight,
  torch::Tensor output,
  torch::Tensor threshold,
  int nr_xnor_gates,
  int nr_additional_samples,
  int majv_shift,
  int threshold_scaling
) {
  // The number of thread blocks in a grid is usually dictated by the size of the data being processed, which typically exceeds the number of processors in the system.
  // dim3 threadsPerBlock(8,8,8)
  // <<<number of blocks per grid, number of threads ber block>>>
  // grid is created with enough blocks to have one thread per matrix element

  // https://devtalk.nvidia.com/default/topic/1028226/how-many-concurrent-threads-are-running-on-my-geforce-gtx-1080-ti-/
  const int output_size_x = output.size(1);
  const int output_size_y = output.size(0);
  const int output_size_z = output.size(2);
  int threads_x = 8; // per block, 8
  int threads_y = 8; // per block, 8
  int threads_z = 8; // per block, 8

  #if DEBUG_1D
    threads_x = 1;
    threads_y = 1;
    threads_z = 1;
  #endif

  const dim3 threads(threads_x, threads_y, threads_z);
  const dim3 blocks((output_size_x + threads_x - 1) / threads_x,
                    (output_size_y + threads_y - 1) / threads_y,
                    (output_size_z + threads_z - 1) / threads_z);

  AT_DISPATCH_ALL_TYPES(input.type(), "customconv2d_cuda", ([&] {
    customconv2d_kernel<scalar_t><<<blocks, threads>>>(
        input.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
        weight.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        output.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
        threshold.packed_accessor<scalar_t,1,torch::RestrictPtrTraits,size_t>(),
        nr_xnor_gates,
        nr_additional_samples,
        majv_shift,
        threshold_scaling
    );
  }));

  return output;
}
